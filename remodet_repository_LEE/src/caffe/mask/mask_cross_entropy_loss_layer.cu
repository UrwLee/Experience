#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/mask/mask_cross_entropy_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MaskCrossEntropyLossBPKernel(const int nthreads, const Dtype thre_min, const Dtype thre_max,
          const Dtype* bottom_data, const Dtype* bottom_label, const Dtype* flags, const int offsize,
          const Dtype scale, Dtype* diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int id = index / offsize;
    int flag = flags[id];
    if (flag > 0) {
      Dtype pred = bottom_data[index];
      pred = min(max(pred,thre_min),thre_max);
      diff[index] = scale * (bottom_label[index] / pred - (1.0 - bottom_label[index]) / (1.0 - pred));
    } else {
      diff[index] = 0;
    }
  }
}

template <typename Dtype>
void MaskCrossEntropyLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  // if (propagate_down[1]) {
  //   LOG(FATAL) << this->type()
  //              << " Layer cannot backpropagate to label inputs.";
  // }
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* bottom_label = bottom[1]->gpu_data();
    const Dtype* flags = bottom[1]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    int num = bottom[0]->num();
    int channels = bottom[0]->channels();
    int offsize = bottom[0]->count() / num / channels;
    Dtype scale_temp = this->layer_param_.mask_loss_param().scale();
    const Dtype scale = - top[0]->cpu_diff()[0] / num * scale_temp;
    const int count = bottom[0]->count();
    MaskCrossEntropyLossBPKernel<Dtype><<<CAFFE_GET_BLOCKS(count),CAFFE_CUDA_NUM_THREADS>>>(
      count, kLOG_THRESHOLD, 1.0 - kLOG_THRESHOLD, bottom_data, bottom_label, flags, offsize,
      scale, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_BACKWARD(MaskCrossEntropyLossLayer);
}
