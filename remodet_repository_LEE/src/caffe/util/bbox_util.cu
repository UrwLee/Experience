#include "hip/hip_runtime.h"
#include <algorithm>
#include <functional>
#include <map>
#include <vector>

#include "thrust/functional.h"
#include "thrust/sort.h"

#include "caffe/common.hpp"
#include "caffe/util/bbox_util.hpp"

namespace caffe {

template <typename Dtype>
__host__ __device__ Dtype BBoxSizeGPU(const Dtype* bbox,
    const bool normalized) {
  if (bbox[2] < bbox[0] || bbox[3] < bbox[1]) {
    // If bbox is invalid (e.g. xmax < xmin or ymax < ymin), return 0.
    return Dtype(0.);
  } else {
    const Dtype width = bbox[2] - bbox[0];
    const Dtype height = bbox[3] - bbox[1];
    if (normalized) {
      return width * height;
    } else {
      // If bbox is not within range [0, 1].
      return (width + 1) * (height + 1);
    }
  }
}

template __host__ __device__ float BBoxSizeGPU(const float* bbox,
    const bool normalized);
template __host__ __device__ double BBoxSizeGPU(const double* bbox,
    const bool normalized);

template <typename Dtype>
__host__ __device__ Dtype JaccardOverlapGPU(const Dtype* bbox1,
    const Dtype* bbox2) {
  if (bbox2[0] > bbox1[2] || bbox2[2] < bbox1[0] ||
      bbox2[1] > bbox1[3] || bbox2[3] < bbox1[1]) {
    return Dtype(0.);
  } else {
    const Dtype inter_xmin = max(bbox1[0], bbox2[0]);
    const Dtype inter_ymin = max(bbox1[1], bbox2[1]);
    const Dtype inter_xmax = min(bbox1[2], bbox2[2]);
    const Dtype inter_ymax = min(bbox1[3], bbox2[3]);

    const Dtype inter_width = inter_xmax - inter_xmin;
    const Dtype inter_height = inter_ymax - inter_ymin;
    const Dtype inter_size = inter_width * inter_height;

    const Dtype bbox1_size = BBoxSizeGPU(bbox1);
    const Dtype bbox2_size = BBoxSizeGPU(bbox2);

    return inter_size / (bbox1_size + bbox2_size - inter_size);
  }
}

template __host__ __device__ float JaccardOverlapGPU(const float* bbox1,
    const float* bbox2);
template __host__ __device__ double JaccardOverlapGPU(const double* bbox1,
    const double* bbox2);

template <typename Dtype>
__device__ Dtype Min(const Dtype x, const Dtype y) {
  return x < y ? x : y;
}

template <typename Dtype>
__device__ Dtype Max(const Dtype x, const Dtype y) {
  return x > y ? x : y;
}

template <typename Dtype>
__device__ void ClipBBoxGPU(const Dtype* bbox, Dtype* clip_bbox) {
  for (int i = 0; i < 4; ++i) {
    clip_bbox[i] = Max(Min(bbox[i], Dtype(1.)), Dtype(0.));
  }
}

template __device__ void ClipBBoxGPU(const float* bbox, float* clip_bbox);
template __device__ void ClipBBoxGPU(const double* bbox, double* clip_bbox);

template <typename Dtype>
__global__ void DecodeBBoxesKernel(const int nthreads,
          const Dtype* loc_data, const Dtype* prior_data,
          const CodeType code_type, const bool variance_encoded_in_target,
          const int num_priors, Dtype* bbox_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int i = index % 4;
    const int d = (index / 4) % num_priors;
    const int pi = d * 4;
    const int vi = pi + num_priors * 4;
    if (code_type == PriorBoxParameter_CodeType_CORNER) {
      if (variance_encoded_in_target) {
        // variance is encoded in target, we simply need to add the offset
        // predictions.
        bbox_data[index] = prior_data[pi + i] + loc_data[index];
      } else {
        // variance is encoded in bbox, we need to scale the offset accordingly.
        bbox_data[index] =
          prior_data[pi + i] + loc_data[index] * prior_data[vi + i];
      }
    } else if (code_type == PriorBoxParameter_CodeType_CENTER_SIZE) {
      const Dtype p_xmin = prior_data[pi];
      const Dtype p_ymin = prior_data[pi + 1];
      const Dtype p_xmax = prior_data[pi + 2];
      const Dtype p_ymax = prior_data[pi + 3];
      const Dtype prior_width = p_xmax - p_xmin;
      const Dtype prior_height = p_ymax - p_ymin;
      const Dtype prior_center_x = (p_xmin + p_xmax) / 2.;
      const Dtype prior_center_y = (p_ymin + p_ymax) / 2.;

      const Dtype xmin = loc_data[index - i];
      const Dtype ymin = loc_data[index - i + 1];
      const Dtype xmax = loc_data[index - i + 2];
      const Dtype ymax = loc_data[index - i + 3];

      Dtype decode_bbox_center_x, decode_bbox_center_y;
      Dtype decode_bbox_width, decode_bbox_height;
      if (variance_encoded_in_target) {
        // variance is encoded in target, we simply need to retore the offset
        // predictions.
        decode_bbox_center_x = xmin * prior_width + prior_center_x;
        decode_bbox_center_y = ymin * prior_height + prior_center_y;
        decode_bbox_width = exp(xmax) * prior_width;
        decode_bbox_height = exp(ymax) * prior_height;
      } else {
        // variance is encoded in bbox, we need to scale the offset accordingly.
        decode_bbox_center_x =
          prior_data[vi] * xmin * prior_width + prior_center_x;
        decode_bbox_center_y =
          prior_data[vi + 1] * ymin * prior_height + prior_center_y;
        decode_bbox_width =
          exp(prior_data[vi + 2] * xmax) * prior_width;
        decode_bbox_height =
          exp(prior_data[vi + 3] * ymax) * prior_height;
      }

      switch (i) {
        case 0:
          bbox_data[index] = decode_bbox_center_x - decode_bbox_width / 2.;
          break;
        case 1:
          bbox_data[index] = decode_bbox_center_y - decode_bbox_height / 2.;
          break;
        case 2:
          bbox_data[index] = decode_bbox_center_x + decode_bbox_width / 2.;
          break;
        case 3:
          bbox_data[index] = decode_bbox_center_y + decode_bbox_height / 2.;
          break;
      }
    } else {
      // Unknown code type.
    }
  }
}

template <typename Dtype>
void DecodeBBoxesGPU(const int nthreads,
          const Dtype* loc_data, const Dtype* prior_data,
          const CodeType code_type, const bool variance_encoded_in_target,
          const int num_priors, Dtype* bbox_data) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  DecodeBBoxesKernel<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, loc_data, prior_data, code_type,
      variance_encoded_in_target, num_priors, bbox_data);
  CUDA_POST_KERNEL_CHECK;
}

template void DecodeBBoxesGPU(const int nthreads,
          const float* loc_data, const float* prior_data,
          const CodeType code_type, const bool variance_encoded_in_target,
          const int num_priors, float* bbox_data);
template void DecodeBBoxesGPU(const int nthreads,
          const double* loc_data, const double* prior_data,
          const CodeType code_type, const bool variance_encoded_in_target,
          const int num_priors, double* bbox_data);

template <typename Dtype>
__global__ void DecodeBBoxesByLocKernel(const int nthreads,
          const Dtype* loc_data, const Dtype* prior_data,
          const int w, const int h, const int num_priors,
          Dtype* bbox_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int l = index % 4;
    int temp = index / 4;
    const int n = temp % num_priors;
    temp /= num_priors;
    const int j = temp % w;
    temp /= w;
    const int i = temp % h;

    const Dtype pw = prior_data[2*n];
    const Dtype ph = prior_data[2*n+1];

    const Dtype pred_cx = loc_data[index - l];
    const Dtype pred_cy = loc_data[index - l + 1];
    const Dtype pred_w = loc_data[index - l + 2];
    const Dtype pred_h = loc_data[index - l + 3];

    Dtype center_x, center_y, width, height;
    center_x = (j + 1. / (1. + exp(-pred_cx))) / w;
    center_y = (i + 1. / (1. + exp(-pred_cy))) / h;
    width = exp(pred_w) * pw;
    height = exp(pred_h) * ph;

    switch (l) {
      case 0:
        bbox_data[index] = center_x - width / 2.;
        break;
      case 1:
        bbox_data[index] = center_y - height / 2.;
        break;
      case 2:
        bbox_data[index] = center_x + width / 2.;
        break;
      case 3:
        bbox_data[index] = center_y + height / 2.;
        break;
    }
  }
}

template <typename Dtype>
void DecodeBBoxesByLocGPU(const int nthreads,
          const Dtype* loc_data, const Dtype* prior_data,
          const int w, const int h, const int num_priors,
          Dtype* bbox_data) {
  DecodeBBoxesByLocKernel<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, loc_data, prior_data,
      w, h, num_priors, bbox_data);
  CUDA_POST_KERNEL_CHECK;
}

template void DecodeBBoxesByLocGPU(const int nthreads,
          const float* loc_data, const float* prior_data,
          const int w, const int h, const int num_priors,
          float* bbox_data);
template void DecodeBBoxesByLocGPU(const int nthreads,
          const double* loc_data, const double* prior_data,
          const int w, const int h, const int num_priors,
          double* bbox_data);

template <typename Dtype>
__global__ void PermuteDataKernel(const int nthreads,
          const Dtype* data, const int num_classes, const int num_data,
          const int num_dim, Dtype* new_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int i = index % num_dim;
    const int c = (index / num_dim) % num_classes;
    const int d = (index / num_dim / num_classes) % num_data;
    const int n = index / num_dim / num_classes / num_data;
    const int new_index = ((n * num_classes + c) * num_data + d) * num_dim + i;
    new_data[new_index] = data[index];
  }
}

template <typename Dtype>
void PermuteDataGPU(const int nthreads,
          const Dtype* data, const int num_classes, const int num_data,
          const int num_dim, Dtype* new_data) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  PermuteDataKernel<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, data, num_classes, num_data,
      num_dim, new_data);
  CUDA_POST_KERNEL_CHECK;
}

template void PermuteDataGPU(const int nthreads,
          const float* data, const int num_classes, const int num_data,
          const int num_dim, float* new_data);
template void PermuteDataGPU(const int nthreads,
          const double* data, const int num_classes, const int num_data,
          const int num_dim, double* new_data);

// permute
template <typename Dtype>
__global__ void PermuteConfDataToBgClassKernel(const int nthreads,
                    const Dtype* conf_data, const int num_classes,
                    Dtype* bg_data, Dtype* class_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int c = index % (num_classes + 1);
    const int box_idx = index / (num_classes + 1);

    if (c == 0) {
      // push to bg_data
      bg_data[box_idx] = conf_data[index];
    } else {
      // push to class_data
      const int class_idx = box_idx * num_classes + c - 1;
      class_data[class_idx] = conf_data[index];
    }
  }
}

template <typename Dtype>
void PermuteConfDataToBgClassGPU(const int nthreads,
                    const Dtype* conf_data, const int num_classes,
                    Dtype* bg_data, Dtype* class_data) {
  PermuteConfDataToBgClassKernel<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads,conf_data,
      num_classes,bg_data,class_data);
}

template void PermuteConfDataToBgClassGPU(const int nthreads,
                    const float* conf_data, const int num_classes,
                    float* bg_data, float* class_data);
template void PermuteConfDataToBgClassGPU(const int nthreads,
                    const double* conf_data, const int num_classes,
                    double* bg_data, double* class_data);
// logistic
template <typename Dtype>
__global__ void kernel_logistic(const int count, Dtype* data) {
  CUDA_KERNEL_LOOP(index, count) {
    data[index] = 1. / (1. + exp(-data[index]));
  }
}

template <typename Dtype>
void LogisticGPU(const int count, Dtype* data) {
  kernel_logistic<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, data);
}

template void LogisticGPU(const int count, float* data);
template void LogisticGPU(const int count, double* data);

// Max
template <typename Dtype>
__global__ void kernel_channel_max(const int out_dim,
    const int channels, const Dtype* data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, out_dim) {
    Dtype maxval = -FLT_MAX;
    for (int c = 0; c < channels; ++c) {
      maxval = max(data[index * channels + c], maxval);
    }
    out[index] = maxval;
  }
}

// Sub
template <typename Dtype>
__global__ void kernel_channel_subtract(const int count,
    const int channels, const Dtype* channel_data,
    const Dtype* channel_max, Dtype* data) {
  CUDA_KERNEL_LOOP(index, count) {
    int n = index / channels;
    data[index] = channel_data[index] - channel_max[n];
  }
}

// exp
template <typename Dtype>
__global__ void kernel_exp(const int count, Dtype* data) {
  CUDA_KERNEL_LOOP(index, count) {
    data[index] = exp(data[index]);
  }
}

// sum
template <typename Dtype>
__global__ void kernel_channel_sum(const int out_dim,
            const int channels, const Dtype* data,
            Dtype* channel_sum) {
  CUDA_KERNEL_LOOP(index, out_dim) {
    Dtype sum = 0;
    for (int c = 0; c < channels; ++c) {
      sum += data[index * channels + c];
    }
    channel_sum[index] = sum;
  }
}

// div
template <typename Dtype>
__global__ void kernel_channel_div(const int count,
      const int channels, const Dtype* channel_sum,
      Dtype* data) {
  CUDA_KERNEL_LOOP(index, count) {
    int n = index / channels;
    data[index] /= channel_sum[n];
  }
}

// SoftMaxGPU
template <typename Dtype>
void SoftMaxGPU(const Dtype* data, const int out_num,
    const int channels, Dtype* prob) {
  vector<int> shape(1,1);
  shape[0] = out_num;
  Blob<Dtype> scale(shape);
  Dtype* scale_data = scale.mutable_gpu_data();
  const int count = out_num * channels;
  kernel_channel_max<Dtype><<<CAFFE_GET_BLOCKS(out_num),
        CAFFE_CUDA_NUM_THREADS>>>(out_num, channels, data, scale_data);
  kernel_channel_subtract<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(count, channels, data, scale_data, prob);
  kernel_exp<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, prob);
  kernel_channel_sum<Dtype><<<CAFFE_GET_BLOCKS(out_num),
        CAFFE_CUDA_NUM_THREADS>>>(out_num, channels, prob, scale_data);
  kernel_channel_div<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(count, channels, scale_data, prob);
}

template void SoftMaxGPU(const float* data, const int out_num,
    const int channels, float* prob);
template void SoftMaxGPU(const double* data, const int out_num,
    const int channels, double* prob);

template <typename Dtype>
__global__ void kernel_channel_multiply(const int count,
      const int channels, const Dtype* channel_scalar,
      Dtype* data) {
  CUDA_KERNEL_LOOP(index, count) {
    int n = index / channels;
    data[index] *= channel_scalar[n];
  }
}

template <typename Dtype>
void UpdateConfByObjGPU(const int nthreads, const int num_classes,
      const Dtype* objectness, Dtype* conf_data) {
  kernel_channel_multiply<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, num_classes,
        objectness, conf_data);
}

template void UpdateConfByObjGPU(const int nthreads,
  const int num_classes, const float* objectness, float* conf_data);
template void UpdateConfByObjGPU(const int nthreads,
  const int num_classes, const double* objectness, double* conf_data);

template <typename Dtype>
__global__ void ComputeOverlappedKernel(const int nthreads,
          const Dtype* bbox_data, const int num_bboxes, const int num_classes,
          const Dtype overlap_threshold, bool* overlapped_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int j = index % num_bboxes;
    const int i = (index / num_bboxes) % num_bboxes;
    if (i == j) {
      // Ignore same bbox.
      return;
    }
    const int c = (index / num_bboxes / num_bboxes) % num_classes;
    const int n = index / num_bboxes / num_bboxes / num_classes;
    // Compute overlap between i-th bbox and j-th bbox.
    const int start_loc_i = ((n * num_bboxes + i) * num_classes + c) * 4;
    const int start_loc_j = ((n * num_bboxes + j) * num_classes + c) * 4;
    const Dtype overlap = JaccardOverlapGPU<Dtype>(bbox_data + start_loc_i,
        bbox_data + start_loc_j);
    if (overlap > overlap_threshold) {
      overlapped_data[index] = true;
    }
  }
}

template <typename Dtype>
void ComputeOverlappedGPU(const int nthreads,
          const Dtype* bbox_data, const int num_bboxes, const int num_classes,
          const Dtype overlap_threshold, bool* overlapped_data) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  ComputeOverlappedKernel<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, bbox_data, num_bboxes, num_classes,
      overlap_threshold, overlapped_data);
  CUDA_POST_KERNEL_CHECK;
}

template void ComputeOverlappedGPU(const int nthreads,
          const float* bbox_data, const int num_bboxes, const int num_classes,
          const float overlap_threshold, bool* overlapped_data);
template void ComputeOverlappedGPU(const int nthreads,
          const double* bbox_data, const int num_bboxes, const int num_classes,
          const double overlap_threshold, bool* overlapped_data);

template <typename Dtype>
__global__ void ComputeOverlappedByIdxKernel(const int nthreads,
          const Dtype* bbox_data, const Dtype overlap_threshold,
          const int* idx, const int num_idx, bool* overlapped_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int j = index % num_idx;
    const int i = (index / num_idx);
    if (i == j) {
      // Ignore same bbox.
      return;
    }
    // Compute overlap between i-th bbox and j-th bbox.
    const int start_loc_i = idx[i] * 4;
    const int start_loc_j = idx[j] * 4;
    const Dtype overlap = JaccardOverlapGPU<Dtype>(bbox_data + start_loc_i,
        bbox_data + start_loc_j);
    if (overlap > overlap_threshold) {
      overlapped_data[index] = true;
    }
  }
}

template <typename Dtype>
void ComputeOverlappedByIdxGPU(const int nthreads,
          const Dtype* bbox_data, const Dtype overlap_threshold,
          const int* idx, const int num_idx, bool* overlapped_data) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  ComputeOverlappedByIdxKernel<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, bbox_data, overlap_threshold,
      idx, num_idx, overlapped_data);
  CUDA_POST_KERNEL_CHECK;
}

template void ComputeOverlappedByIdxGPU(const int nthreads,
          const float* bbox_data, const float overlap_threshold,
          const int* idx, const int num_idx, bool* overlapped_data);
template void ComputeOverlappedByIdxGPU(const int nthreads,
          const double* bbox_data, const double overlap_threshold,
          const int* idx, const int num_idx, bool* overlapped_data);

template <typename Dtype>
void ApplyNMSGPU(const Dtype* bbox_data, const Dtype* conf_data,
          const int num_bboxes, const float confidence_threshold,
          const int top_k, const float nms_threshold, vector<int>* indices) {
  // Keep part of detections whose scores are higher than confidence threshold.
  vector<int> idx;
  vector<Dtype> confidences;
  for (int i = 0; i < num_bboxes; ++i) {
    if (conf_data[i] > confidence_threshold) {
      idx.push_back(i);
      confidences.push_back(conf_data[i]);
    }
  }
  int num_remain = confidences.size();
  if (num_remain == 0) {
    return;
  }
  // Sort detections based on score.
  thrust::sort_by_key(&confidences[0], &confidences[0] + num_remain, &idx[0],
      thrust::greater<Dtype>());
  if (top_k > -1 && top_k < num_remain) {
    num_remain = top_k;
  }

  // Compute overlap between remaining detections.
  Blob<int> idx_blob(1, 1, 1, num_remain);
  int* idx_data = idx_blob.mutable_cpu_data();
  std::copy(idx.begin(), idx.begin() + num_remain, idx_data);

  Blob<bool> overlapped(1, 1, num_remain, num_remain);
  const int total_bboxes = overlapped.count();
  bool* overlapped_data = overlapped.mutable_gpu_data();
  ComputeOverlappedByIdxGPU<Dtype>(total_bboxes, bbox_data, nms_threshold,
      idx_blob.gpu_data(), num_remain, overlapped_data);

  // Do non-maximum suppression based on overlapped results.
  const bool* overlapped_results = overlapped.cpu_data();
  vector<int> selected_indices;
  ApplyNMS(overlapped_results, num_remain, &selected_indices);

  // Put back the selected information.
  for (int i = 0; i < selected_indices.size(); ++i) {
    indices->push_back(idx[selected_indices[i]]);
  }
}

template
void ApplyNMSGPU(const float* bbox_data, const float* conf_data,
          const int num_bboxes, const float confidence_threshold,
          const int top_k, const float nms_threshold, vector<int>* indices);
template
void ApplyNMSGPU(const double* bbox_data, const double* conf_data,
          const int num_bboxes, const float confidence_threshold,
          const int top_k, const float nms_threshold, vector<int>* indices);

template <typename Dtype>
__global__ void GetDetectionsKernel(const int nthreads,
          const Dtype* bbox_data, const Dtype* conf_data, const int image_id,
          const int label, const int* indices, const bool clip_bbox,
          Dtype* detection_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int det_idx = indices[index];
    detection_data[index * 7] = image_id;
    detection_data[index * 7 + 1] = label;
    detection_data[index * 7 + 2] = conf_data[det_idx];
    if (clip_bbox) {
      ClipBBoxGPU(&(bbox_data[det_idx * 4]), &(detection_data[index * 7 + 3]));
    } else {
      for (int i = 0; i < 4; ++i) {
        detection_data[index * 7 + 3 + i] = bbox_data[det_idx * 4 + i];
      }
    }
  }
}

template <typename Dtype>
void GetDetectionsGPU(const Dtype* bbox_data, const Dtype* conf_data,
          const int image_id, const int label, const vector<int>& indices,
          const bool clip_bbox, Blob<Dtype>* detection_blob) {
  // Store selected indices in array.
  int num_det = indices.size();
  if (num_det == 0) {
    return;
  }
  Blob<int> idx_blob(1, 1, 1, num_det);
  int* idx_data = idx_blob.mutable_cpu_data();
  std::copy(indices.begin(), indices.end(), idx_data);
  // Prepare detection_blob.
  detection_blob->Reshape(1, 1, num_det, 7);
  Dtype* detection_data = detection_blob->mutable_gpu_data();
  // NOLINT_NEXT_LINE(whitespace/operators)
  GetDetectionsKernel<Dtype><<<CAFFE_GET_BLOCKS(num_det),
      CAFFE_CUDA_NUM_THREADS>>>(num_det, bbox_data, conf_data, image_id, label,
      idx_blob.gpu_data(), clip_bbox, detection_data);
  CUDA_POST_KERNEL_CHECK;
}

template void GetDetectionsGPU(const float* bbox_data, const float* conf_data,
          const int image_id, const int label, const vector<int>& indices,
          const bool clip_bbox, Blob<float>* detection_blob);
template void GetDetectionsGPU(const double* bbox_data, const double* conf_data,
          const int image_id, const int label, const vector<int>& indices,
          const bool clip_bbox, Blob<double>* detection_blob);

}  // namespace caffe
